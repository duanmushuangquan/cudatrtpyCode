#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>


__global__ void compute(float* a, float* b, float* c){
    int d0 = gridDim.z;
    int d1 = gridDim.y;
    int d2 = gridDim.x;
    int d3 = blockDim.z;
    int d4 = blockDim.y;
    int d5 = blockDim.x;

    int p0 = blockIdx.z;
    int p1 = blockIdx.y;
    int p2 = blockIdx.x;
    int p3 = threadIdx.z;
    int p4 = threadIdx.y;
    int p5 = threadIdx.x;

    int position = (((((p0 * d1) + p1) * d2 + p2) * d3 + p3) * d4 + p4) * d5 + p5;
    c[position] = a[position] * b[position];

    printf("gridDim=%dx%dx%dx, blockDim = %dx%dx%d, [blockIdx = %d,%d,%d, threadIdx = %d,%d,%d], position = %d, avalue = %f\n",
            gridDim.x, gridDim.y, gridDim.z,
            blockDim.x, blockDim.y, blockDim.z,
            blockIdx.x, blockIdx.y, blockIdx.z,
            threadIdx.x, threadIdx.y, threadIdx.z,
            position, a[position]);
}

int main(){
    hipProfilerStart();

    const int num = 16;
    float a[num] = {1, 2, 3};
    float b[num] = {5, 7, 9};
    float c[num] = {0};

    for(int i=0; i<num;++i){
        a[i] = i;
        b[i] = i;
    }
    size_t size_array = sizeof(c);
    float* device_a = nullptr;
    float* device_b = nullptr;
    float* device_c = nullptr;


    hipMalloc(&device_a, size_array);
    hipMalloc(&device_b, size_array);
    hipMalloc(&device_c, size_array);

    hipMemcpy(device_a, a, size_array, hipMemcpyHostToDevice);
    hipMemcpy(device_b, b, size_array, hipMemcpyHostToDevice);

    compute<<<dim3(1, 2, 2), dim3(2, 2)>>>(device_a, device_b, device_c);

    hipMemcpy(c, device_c, size_array, hipMemcpyDeviceToHost);
    for(int i = 0; i< 16;++i){
        printf("c[%d] = %f\n", i, c[i]);
    }

    return 0;
}
